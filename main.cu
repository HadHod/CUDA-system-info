#include <iostream>

using namespace std;

int main() {
    int ver;
    hipDriverGetVersion(&ver);
    cout << "DRIVER VERSION: " << ver << "\n";
    hipRuntimeGetVersion (&ver);
    cout << "RUNTIME VERSION: " << ver << "\n";

    cout << "\n";

    size_t total_mem, free_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    cout << "TOTAL MEMORY: " << total_mem * 1e-6f << "MB\n";
    cout << "FREE MEMORY: " << free_mem * 1e-6f << "MB\n";

    cout << "\n";

    hipDeviceProp_t prop;
    int numberOfDevices;

    hipGetDeviceCount(&numberOfDevices);
    cout << "Number of devices: " << numberOfDevices << "\n";

    for (int i = 0; i < numberOfDevices; i++) {
        cout << "\n";
        hipGetDeviceProperties(&prop, i);

        cout << "Name: " << prop.name << endl;

        cout << "Clock rate: " << prop.clockRate * 1e-6f << "Ghz\n";
        cout << "Device compute capability: " << prop.major << "." << prop.minor << "\n";
        cout << "Multiprocessor count: " << prop.multiProcessorCount << "\n";
        cout << "Total global mem: " << prop.totalGlobalMem * 1e-6f << "MB\n";
        cout << "Max threads per MProcessor: " << prop.maxThreadsPerMultiProcessor << "\n";
        cout << "Max threads pen Block: " << prop.maxThreadsPerBlock << "\n";
        cout << "Shared memory per Block: " << prop.sharedMemPerBlock * 1e-6f << "MB\n";
        cout << "Warp size: " << prop.warpSize << "\n";
    }

    cout << endl;

    return 0;
}