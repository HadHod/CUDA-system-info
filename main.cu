#include <iostream>

using namespace std;

int main() {
    int ver;
    hipDriverGetVersion(&ver);
    cout << "DRIVER VERSION: " << ver << "\n";
    hipRuntimeGetVersion (&ver);
    cout << "RUNTIME VERSION: " << ver << "\n";

    cout << "\n";

    size_t total_mem, free_mem;
    hipMemGetInfo(&free_mem, &total_mem);
    cout << "TOTAL MEMORY: " << total_mem * 1e-6f << "MB\n";
    cout << "FREE MEMORY: " << free_mem * 1e-6f << "MB\n";

    cout << "\n";

    hipDeviceProp_t prop;
    int numberOfDevices;

    hipGetDeviceCount(&numberOfDevices);
    cout << "Number of devices: " << numberOfDevices << "\n";

    for (int i = 0; i < numberOfDevices; i++) {
        cout << "\n";
        hipGetDeviceProperties(&prop, i);

        cout << "Name: " << prop.name << endl;

        cout << "Clock rate: " << prop.clockRate * 1e-6f << "Ghz\n";
        cout << "Device compute capability: " << prop.major << "." << prop.minor << "\n";
        cout << "Multiprocessor count: " << prop.multiProcessorCount << "\n";
        cout << "Total global mem: " << prop.totalGlobalMem * 1e-6f << "MB\n";
        cout << "Max threads per MProcessor: " << prop.maxThreadsPerMultiProcessor << "\n";
        cout << "Shared memory per Block: " << prop.sharedMemPerBlock * 1e-6f << "MB\n";
        cout << "Warp size: " << prop.warpSize << "\n";
        cout << "Max threads pen Block: " << prop.maxThreadsPerBlock << "\n";
        cout << "Max block dimension: (" << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")\n";
        cout << "Max grid dimension: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")\n";
        cout << "Kernel execution timeout is " << (prop.kernelExecTimeoutEnabled ? "on" : "off") << "\n";
        cout << "Device overlap is " << (prop.deviceOverlap ? "on" : "off") << "\n";
    }

    cout << endl;

    return 0;
}